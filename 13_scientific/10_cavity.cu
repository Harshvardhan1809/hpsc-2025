
#include <hip/hip_runtime.h>
#include <cstdlib>
#include <cstdio>
#include <fstream>

using namespace std;

__managed__ int nx = 41;
__managed__ int ny=41;
__managed__ int nt = 500;
__managed__ int nit = 50;
__managed__ double dx;
__managed__ double dy;
__managed__ double dt = .01;
__managed__ double rho = 1.;
__managed__ double nu = .02;

__global__ void u_v_p_b_init(float *u, float *v, float *p, float *b){
    u[threadIdx.x] = 0;
    v[threadIdx.x] = 0;
    p[threadIdx.x] = 0;
    b[threadIdx.x] = 0;
}

__global__ void b_calc(float *b, float *u, float *v){
    if(threadIdx.x/ny == 0 || threadIdx.x%ny == 0 || threadIdx.x%ny == ny-1 || threadIdx.x/ny == nx-1) return ;
    int x = threadIdx.x;
    b[x] = rho * (1/dt*((b[x+1] - b[x-1])/(2*dx) + (v[x+ny] - v[x-ny])/(2*dy)) - ((u[x+1] - u[x-1])/(2*dx))*((u[x+1] - u[x-1])/(2*dx)) - 2*((u[x+ny] - u[x-ny])/(2*dy)*(v[x+1] - v[x-1])/(2*dx)) - ((v[x+ny] - v[x-ny])/(2*dy))*((v[x+ny] - v[x-ny])/(2*dy)));
}

__global__ void pn_p_copy(float *pn, float *p){
    pn[threadIdx.x] = p[threadIdx.x];
}

__global__ void p_calc(float *p, float *pn, float *b){
    if(threadIdx.x/ny == 0 || threadIdx.x%ny == 0 || threadIdx.x%ny == ny-1 || threadIdx.x/ny == nx-1) return ;
    int x = threadIdx.x;
    p[x] = dy*dy*(pn[x+1] + pn[x-1]) + dx*dx*(pn[x+ny] + pn[x-ny]) - b[x]*dx*dx*dy*dy/(2*dx*dx + 2*dy+dy);
}

__global__ void p_op_1(float *p){
    p[threadIdx.x + nx - 1] = p[threadIdx.x + nx -2];
    p[threadIdx.x] = p[threadIdx.x + 1];
}

__global__ void p_op_2(float *p){
    p[(ny-1)*ny + threadIdx.x] = 0;
    p[threadIdx.x] = p[ny + threadIdx.x];
}

__global__ void un_u_vn_v_copy(float *un, float *u, float *vn, float *v){
    un[threadIdx.x] = u[threadIdx.x];
    vn[threadIdx.x] = v[threadIdx.x];
}

__global__ void u_v_calc(float *u, float *v, float *un, float *vn, float *p){
    if(threadIdx.x/ny == 0 || threadIdx.x%ny == 0 || threadIdx.x%ny == ny-1 || threadIdx.x/ny == nx-1) return ;
    int x = threadIdx.x;
    u[x] = un[x] - un[x]*dt/dx*(un[x] - un[x-1]) - un[x]*dt/dy*(un[x] - un[x-ny]) - dt/(2*rho*dx)*(p[x+1] - p[x-1])
            + nu*dt/(dx*dx)*(un[x+1] - 2*un[x] + un[x-1]) + nu*dt/(dy*dy)*(un[x+ny] - 2*un[x] + un[x-ny]);
    v[x] = vn[x] - vn[x]*dt/dx*(vn[x] - vn[x-1]) - vn[x]*dt/dy*(vn[x] - vn[x-ny]) - dt/(2*rho*dx)*(p[x+1] - p[x-1])
            + nu*dt/(dx*dx)*(vn[x+1] - 2*vn[x] + vn[x-1]) + nu*dt/(dy*dy)*(vn[x+ny] - 2*vn[x] + vn[x-ny]);
}

__global__ void u_v_op_1(float *u, float *v){
    u[threadIdx.x*ny] = 0;
    v[threadIdx.x*ny] = 0;
    u[threadIdx.x*ny + nx-1] = 0;
    v[threadIdx.x*ny + nx-1] = 0;
}

__global__ void u_v_op_2(float *u, float *v){
    u[threadIdx.x] = 0;
    u[(ny-1)*nx + threadIdx.x] = 1;
    v[threadIdx.x] = 0;
    v[(ny-1)*nx + threadIdx.x] = 0;
    
}

int main() {
//   nx = 41;
//   ny = 41;
//   nt = 500;
//   nit = 50;
  dx = 2. / (nx - 1);
  dy = 2. / (ny - 1);
//   dt = .01;
//   rho = 1.;
//   nu = .02;

  float *u, *v, *p, *b, *un, *vn, *pn;
  hipMallocManaged(&u, ny*nx*sizeof(int));
  hipMallocManaged(&v, ny*nx*sizeof(int));
  hipMallocManaged(&p, ny*nx*sizeof(int));
  hipMallocManaged(&b, ny*nx*sizeof(int));
  hipMallocManaged(&un, ny*nx*sizeof(int));
  hipMallocManaged(&vn, ny*nx*sizeof(int));
  hipMallocManaged(&pn, ny*nx*sizeof(int));

  u_v_p_b_init<<<1, nx*ny>>>(u, v, p, b);

  ofstream ufile("u_cu.dat");
  ofstream vfile("v_cu.dat");
  ofstream pfile("p_cu.dat");

  for(int n=0; n<nt; n++){
    b_calc<<<1, nx*ny>>>(b, u, v);

    for(int it=0; it<nit; it++){
        pn_p_copy<<<1, nx*ny>>>(p, pn);
        p_calc<<<1, nx*ny>>>(p, pn, b);
        p_op_1<<<1, ny>>>(p);
        p_op_2<<<1, nx>>>(p);
    }

    un_u_vn_v_copy<<<1, nx*ny>>>(un, u, vn, v);

    u_v_calc<<<1, nx*ny>>>(u, v, un, vn, p);

    u_v_op_1<<<1, ny>>>(u, v);

    u_v_op_2<<<1, nx>>>(u, v);

    if (n % 10 == 0) {
      for (int x=0; x<nx*ny; x++)  ufile << u[x] << " ";
      ufile << "\n";
      for (int x=0; x<nx*ny; x++)  vfile << v[x] << " ";
      vfile << "\n";
      for (int x=0; x<nx*ny; x++)  pfile << p[x] << " ";
      pfile << "\n";
    }
  }

  ufile.close();
  vfile.close();
  pfile.close();
}