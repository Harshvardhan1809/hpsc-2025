
#include <hip/hip_runtime.h>
#include <cstdio>
#include <cstdlib>
#include <vector>

__global__ void  init(int *bucket){
  bucket[threadIdx.x] = 0;
}

__global__ void buck(int *bucket, int *key){
  atomicAdd(&bucket[key[threadIdx.x]], 1);
}

int main() {
  int n = 50;
  int range = 5;

  int *key;
  hipMallocManaged(&key, n*sizeof(int));
  for (int i=0; i<n; i++) {
    key[i] = rand() % range;
    printf("%d ",key[i]);
  }
  printf("\n"); 

  int *bucket;
  hipMallocManaged(&bucket, range*sizeof(int));

  init<<<1,range>>>(bucket);
  hipDeviceSynchronize();

  buck<<<1, n>>>(bucket, key);
  hipDeviceSynchronize();

  // not straight-forward to parallelize; for less iterations serial is okay
  for (int i=0, j=0; i<range; i++) {
    for (; bucket[i]>0; bucket[i]--) {
      key[j++] = i;
    }
  }

  for (int i=0; i<n; i++) {
    printf("%d ",key[i]);
  }
  printf("\n");
}